#include <hiprand.h>
#include "cuda_random.h"

static hiprandGenerator_t rn_gen;
static int NGAUSS=0, NUNIF=0;


extern "C" void init_RNG(int n_gaussnumbers, int n_unifnumbers, int seed){
 
 
  hipError_t cudaerr;
  //determine sizes for gauss numbers
  
  printf("Initializing GPU random number generator...\n");
  hiprandCreateGenerator(&rn_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(rn_gen, seed);
  
  //determine sizes for gaussian numbers
  printf("No. of gauss random numbers: %d\n", n_gaussnumbers );
  NGAUSS = n_gaussnumbers;
   
  //determine sizes for unif. numbers 
  printf("No. of unif. dist. random numbers: %d\n", n_unifnumbers );
  NUNIF = n_unifnumbers;
   

  //allocate fields for random numbers
  printf("Allocating device memory for random numbers...\n");
  CUDA_SAFE_CALL(hipMalloc((void **)&dev_rndgauss_field, n_gaussnumbers * sizeof(float)) );
  CUDA_SAFE_CALL(hipMalloc((void **)&dev_rndunif_field, n_unifnumbers * sizeof(float)));


 // CREATE FIRST RANDOM NUMBERS 
 /* update the random field for gauss numbers*/
    hiprandGenerateNormal(rn_gen, dev_rndgauss_field, n_gaussnumbers, 0.0f, 1.0f);
    
  /* update the random field for unif. dist. numbers*/
    hiprandGenerateUniform(rn_gen, dev_rndunif_field, n_unifnumbers);
   
   cudaerr = hipGetLastError();
   if(cudaerr != hipSuccess){
     printf("%s\n", hipGetErrorString(cudaerr)); 
   } 

}





extern "C" void update_RNG(){


 /* update the random field for gauss numbers*/
    hiprandGenerateNormal(rn_gen, dev_rndgauss_field, NGAUSS, 0.0f, 1.0f);
    
  /* update the random field for unif. dist. numbers*/
    hiprandGenerateUniform(rn_gen, dev_rndunif_field, NUNIF);
    
}



extern "C" void finalize_RNG(){
  hipFree(dev_rndgauss_field);
  hipFree(dev_rndunif_field);
  hiprandDestroyGenerator(rn_gen);
}





